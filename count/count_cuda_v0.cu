
#include <hip/hip_runtime.h>
#include <chrono>
#include <ctime>
#include <iostream>
#include <math.h>
#include <random>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__
void countElem(int n, int find, int *data, int *d_result)
{
  int batch_size = n/blockDim.x;
  int start_index = threadIdx.x * batch_size;

  int batch_count = 0;

  for(int i = start_index; i<start_index+batch_size; i++){
    if(data[i] == find){
      batch_count++;
    }
  }

  d_result[threadIdx.x] = batch_count;
  //atomicAdd(result, batch_count);
}

int countElemCPU(int n, int find, int *data){
  unsigned int total_count = 0;
  for(int i = 0; i<n; i++){
    if(data[i] == find){
      total_count++;
    }
  }
  return total_count;
}

__global__
void initialize(int* data, int N){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  for(int i = idx; i <  N; i += blockDim.x){
    data[i] = 50;
  }
}


int main(void)
{
  auto clock = std::chrono::high_resolution_clock();
  auto start = clock.now();

  int N = 1<<28;
  int blockSize = 256;
  int numBlocks = 1;

  int *d_result = new int[blockSize*numBlocks];
  int *d_data = new int[N];

  // Allocate Unified Memory – accessible from CPU or GPU
  HANDLE_ERROR(hipMallocManaged(&d_data, N*sizeof(int)));
  HANDLE_ERROR(hipMallocManaged(&d_result, blockSize*numBlocks*sizeof(int)));

  initialize<<<8, 256>>>(d_data, N);

  hipMemcpy(d_data, d_data, N * sizeof(int), hipMemcpyHostToDevice);
  
  countElem<<<numBlocks, blockSize>>>(N, 50,d_data, d_result);

  // Wait for GPU to finish before accessing on host
  HANDLE_ERROR(hipDeviceSynchronize());

  HANDLE_ERROR(hipMemcpy(d_result, d_result, blockSize*numBlocks*sizeof(int), hipMemcpyDeviceToHost));

  int final_count = 0;
  for(int i = 0; i<blockSize; i++){
    final_count += d_result[i];
  }

  std::cout << "Element count: " << N << std::endl;  
  std::cout << "Device variable value: " << final_count <<std::endl;

  // Free memory
  HANDLE_ERROR(hipFree(d_result));
  HANDLE_ERROR(hipFree(d_data));

  auto end = clock.now(); 
  long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "Time: "<<  (float) microseconds/1000 << "ms" << std::endl;

  return 0;
}