
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__
void countElem(int n, int find, int *data, int *d_result)
{
  int batch_size = n/gridDim.x;
  int idx = threadIdx.x + batch_size*blockIdx.x;

  int batch_count = 0;

  for(int i = idx; i<batch_size*(blockIdx.x+1); i+=blockDim.x){
    if(data[i] == find){
      batch_count++;
    }
  }
 
  atomicAdd(&d_result[blockIdx.x], batch_count);
}

int countElemCPU(int n, int find, int *data){
  unsigned int total_count = 0;
  for(int i = 0; i<n; i++){
    if(data[i] == find){
      total_count++;
    }
  }
  return total_count;
}

__global__
void initialize(int* data, int N){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;

  for(int i = idx; i <  N; i += blockDim.x){
    data[i] = 50;
  }
}

int main(void)
{

  auto clock = std::chrono::high_resolution_clock();
  auto start = clock.now();

  int N = 1<<28;
  int blockSize = 256;
  int numBlocks = 8;

  int *d_result = new int[numBlocks];
  int *result = new int[numBlocks];
  int *data = new int[N];
  int *d_data = new int[N];

  HANDLE_ERROR(hipMalloc(&d_data, N*sizeof(int)));
  HANDLE_ERROR(hipMalloc(&d_result, numBlocks*sizeof(int)));

  initialize<<<numBlocks, blockSize>>>(d_data, N);
  
  countElem<<<numBlocks, blockSize>>>(N, 50,d_data, d_result);

  HANDLE_ERROR(hipDeviceSynchronize());

  HANDLE_ERROR(hipMemcpy(result, d_result, numBlocks*sizeof(int), hipMemcpyDeviceToHost));

  int final_count = 0;
  for(int i = 0; i<numBlocks; i++){
    final_count += result[i];
  }

  std::cout << "Element count: " << N << std::endl;  
  std::cout << "Device variable value: " << final_count <<std::endl;

  // Free memory
  HANDLE_ERROR(hipFree(d_result));
  HANDLE_ERROR(hipFree(d_data));

  auto end = clock.now(); 
  long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>(end-start).count();
  std::cout << "Time: "<<  (float) microseconds/1000 << "ms" << std::endl;

  return 0;
}